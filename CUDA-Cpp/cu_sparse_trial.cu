#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
//#include <cusparse.h>
#include <hipsparse.h>
#include <iostream>

int main() {
	hipError_t cudaStat1,cudaStat2,cudaStat3,cudaStat4,cudaStat5,cudaStat6;
    hipsparseStatus_t status;
    hipsparseHandle_t handle=0;
    hipsparseMatDescr_t descra=0;
    int *    cooRowIndexHostPtr=0;
    int *    cooColIndexHostPtr=0;    
    double * cooValHostPtr=0;
    int *    cooRowIndex=0;
    int *    cooColIndex=0;    
    double * cooVal=0;
    int *    xIndHostPtr=0;
    double * xValHostPtr=0;
    double * yHostPtr=0;
    int *    xInd=0;
    double * xVal=0;
    double * y=0;  
    int *    csrRowPtr=0;
    double * zHostPtr=0; 
    double * z=0; 
    int      n, nnz, nnz_vector, i, j;

    printf("testing example\n");
    /* create the following sparse test matrix in COO format */
     /* |1.0     2.0 3.0|
       	|    4.0        |
       	|5.0     6.0 7.0|
       	|    8.0     9.0| */

    n=4; nnz=9; 
    cooRowIndexHostPtr = (int *)   malloc(nnz*sizeof(cooRowIndexHostPtr[0])); 
    cooColIndexHostPtr = (int *)   malloc(nnz*sizeof(cooColIndexHostPtr[0])); 
    cooValHostPtr      = (double *)malloc(nnz*sizeof(cooValHostPtr[0])); 

    cooRowIndexHostPtr[0]=0; cooColIndexHostPtr[0]=0; cooValHostPtr[0]=1.0;  
    cooRowIndexHostPtr[1]=0; cooColIndexHostPtr[1]=2; cooValHostPtr[1]=2.0;  
    cooRowIndexHostPtr[2]=0; cooColIndexHostPtr[2]=3; cooValHostPtr[2]=3.0;  
    cooRowIndexHostPtr[3]=1; cooColIndexHostPtr[3]=1; cooValHostPtr[3]=4.0;  
    cooRowIndexHostPtr[4]=2; cooColIndexHostPtr[4]=0; cooValHostPtr[4]=5.0;  
    cooRowIndexHostPtr[5]=2; cooColIndexHostPtr[5]=2; cooValHostPtr[5]=6.0;
    cooRowIndexHostPtr[6]=2; cooColIndexHostPtr[6]=3; cooValHostPtr[6]=7.0;  
    cooRowIndexHostPtr[7]=3; cooColIndexHostPtr[7]=1; cooValHostPtr[7]=8.0;  
    cooRowIndexHostPtr[8]=3; cooColIndexHostPtr[8]=3; cooValHostPtr[8]=9.0;  

    //print the matrix
    printf("Input data:\n");
    for (i=0; i<nnz; i++){        
        printf("cooRowIndexHostPtr[%d]=%d  ",i,cooRowIndexHostPtr[i]);
        printf("cooColIndexHostPtr[%d]=%d  ",i,cooColIndexHostPtr[i]);
        printf("cooValHostPtr[%d]=%f     \n",i,cooValHostPtr[i]);
    }

    status = hipsparseCreate(&handle);
    status = hipsparseCreateMatDescr(&descra);
    hipsparseSetMatType(descra, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descra, HIPSPARSE_INDEX_BASE_ONE);

	hipError_t cudaErrorCode;
    cudaErrorCode = hipMalloc((void**)&csrRowPtr,(n+1)*sizeof(csrRowPtr[0]));
    status = hipsparseXcoo2csr(handle, cooRowIndex,nnz, n, 
    	csrRowPtr, HIPSPARSE_INDEX_BASE_ZERO);

    std::cout << "cuda error code = " << cudaErrorCode << std::endl;
    std::cout << "hipsparseCreate return status = " << status << std::endl;
    return 0;
}